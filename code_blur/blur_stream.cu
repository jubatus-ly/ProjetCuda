#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <opencv2/opencv.hpp>

#include <vector>
#include <chrono>
#include <fstream>

#define BLOCK_SIZE 32
#define RADIUS 5

__global__ void boxBlurFilter(unsigned char* input, unsigned char* output, int width, int height, int radius) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        int sum = 0;
        int count = 0;
        for (int i = -radius; i <= radius; i++) {
            for (int j = -radius; j <= radius; j++) {
                int offsetX = x + i;
                int offsetY = y + j;
                if (offsetX >= 0 && offsetX < width && offsetY >= 0 && offsetY < height) {
                    sum += input[offsetX + offsetY * width];
                    count++;
                }
            }
        }
        output[x + y * width] = sum / count;
    }
}

int main(int argc, char** argv) {
    // Verification de la pr�sence des deux arguments
    if (argc != 3) {
        printf("Usage: edge_detection <input_file> <output_file>\n");
        exit(-1);
    }

    // Charger l'image d'entr�e en niveaux de gris
    cv::Mat input = cv::imread(argv[1], cv::IMREAD_GRAYSCALE);
    if (input.empty()) {
        printf("Unable to load image '%s'\n", argv[1]);
        exit(-1);
    }

    // init cuda error
    hipError_t cudaStatus;
    hipError_t kernelStatus;

    // Allouer la m�moire pour les images d'entr�e et de sortie sur le GPU
    int width = input.cols;
    int height = input.rows;
    int radius = RADIUS;
    size_t inputSize = sizeof(unsigned char) * width * height;
    size_t outputSize = inputSize;
    unsigned char* d_input;
    unsigned char* d_output;
    cudaStatus = hipMalloc(&d_input, inputSize);
    if (cudaStatus != hipSuccess) {
        std::cout << "Error hipMalloc d_input: "  << std::endl;
        exit(-1);
    }
    cudaStatus = hipMalloc(&d_output, outputSize);
    if (cudaStatus != hipSuccess) {
        std::cout << "Error hipMalloc d_output: "  << std::endl;
        exit(-1);
    }

    // Creation des evenement pour le calcul du temps
    hipEvent_t start, stop;
    cudaStatus = hipEventCreate(&start);
    if (cudaStatus != hipSuccess) {
        std::cout << "Error hipEventCreate start: "  << std::endl;
        exit(-1);
    }
    cudaStatus = hipEventCreate(&stop);
    if (cudaStatus != hipSuccess) {
        std::cout << "Error hipEventCreate stop: " << std::endl;
        exit(-1);
    }

    // Mesure du temps de calcul du kernel uniquement
    hipEventRecord( start );

    // Cr�er deux streams CUDA
    hipStream_t stream1, stream2;
    cudaStatus = hipStreamCreate(&stream1);
    if (cudaStatus != hipSuccess) {
        std::cout << "Error hipStreamCreate stream1: "  << std::endl;
        exit(-1);
    }
    cudaStatus = hipStreamCreate(&stream2);
    if (cudaStatus != hipSuccess) {
        std::cout << "Error hipStreamCreate stream2: "  << std::endl;
        exit(-1);
    }

    // D�finir la taille des blocs et des grilles
    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridSize((width + BLOCK_SIZE - 1) / BLOCK_SIZE, (height + BLOCK_SIZE - 1) / BLOCK_SIZE);

    // Copier la premi�re moiti� de l'image d'entr�e sur le GPU avec le premier stream
    cudaStatus = hipMemcpyAsync(d_input, input.data, inputSize / 2, hipMemcpyHostToDevice, stream1);
    if (cudaStatus != hipSuccess) {
        std::cout << "Error hipMemcpyAsync stream1 - HostToDevice: "  << std::endl;
        exit(-1);
    }

    // Copier la deuxi�me moiti� de l'image d'entr�e sur le GPU avec le deuxi�me stream
    cudaStatus = hipMemcpyAsync(d_input + width * (height / 2), input.data + width * (height / 2), inputSize / 2, hipMemcpyHostToDevice, stream2);
    if (cudaStatus != hipSuccess) {
        std::cout << "Error hipMemcpyAsync stream2 - HostToDevice: "  << std::endl;
        exit(-1);
    }

    // Synchroniser les streams
    cudaStatus = hipStreamSynchronize(stream1);
    if (cudaStatus != hipSuccess) {
        std::cout << "Error hipStreamSynchronize stream1: " << std::endl;
        exit(-1);
    }
    cudaStatus = hipStreamSynchronize(stream2);
    if (cudaStatus != hipSuccess) {
        std::cout << "Error hipStreamSynchronize stream2: "  << std::endl;
        exit(-1);
    }

    // Appliquer le filtre de flou sur la premi�re moiti� de l'image d'entr�e avec le premier stream
    boxBlurFilter<<<gridSize, blockSize, 0, stream1>>>(d_input, d_output, width, height / 2, radius);
    kernelStatus = hipGetLastError();
    if (kernelStatus != hipSuccess) {
        std::cout << "CUDA Error: " << hipGetErrorString(kernelStatus) << std::endl;
        exit(-1);
    }

    // Appliquer le filtre de flou sur la deuxi�me moiti� de l'image d'entr�e avec le deuxi�me stream
    boxBlurFilter<<<gridSize, blockSize, 0, stream2>>>(d_input + width * (height / 2), d_output + width * (height / 2), width, height / 2, radius);
    kernelStatus = hipGetLastError();
    if (kernelStatus != hipSuccess) {
        std::cout << "CUDA Error: " << hipGetErrorString(kernelStatus) << std::endl;
        exit(-1);
    }

    // Synchroniser les streams
    cudaStatus = hipStreamSynchronize(stream1);
    if (cudaStatus != hipSuccess) {
        std::cout << "Error hipStreamSynchronize stream1: " << std::endl;
        exit(-1);
    }
    cudaStatus = hipStreamSynchronize(stream2);
    if (cudaStatus != hipSuccess) {
        std::cout << "Error hipStreamSynchronize stream2: "  << std::endl;
        exit(-1);
    }

    // Copier l'image de sortie du GPU vers le CPU avec le premier stream
    cv::Mat output(height, width, CV_8UC1);
    cudaStatus = hipMemcpyAsync(output.data, d_output, outputSize, hipMemcpyDeviceToHost, stream1);
    if (cudaStatus != hipSuccess) {
        std::cout << "Error hipMemcpyAsync stream1 - DeviceToHost: "  << std::endl;
        exit(-1);
    }

    // Fin du temps de calcul
    cudaStatus = hipEventRecord( stop );
    if (cudaStatus != hipSuccess) {
        std::cout << "Error hipEventRecord stop : "  << std::endl;
        exit(-1);
    }

    // Calcul du temps total
    cudaStatus = hipEventSynchronize(stop);
    if (cudaStatus != hipSuccess) {
        std::cout << "Error hipEventSynchronize stop : "  << std::endl;
        exit(-1);
    }
    float duration;
    hipEventElapsedTime( &duration, start, stop );
    std::cout << "time=" << duration << std::endl;

    std::ofstream file("temps.txt", std::ios_base::app);
    file << "blur_stream : " << duration << " ms\n" << std::endl;
    file.close();

    // Enregistrer l'image de sortie
    cv::imwrite(argv[2], output);

    // Lib�rer la m�moire allou�e sur le GPU
    hipFree(d_input);
    hipFree(d_output);
    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    
	return 0;
}