#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <opencv2/opencv.hpp>

#include <vector>
#include <chrono>
#include <fstream>

#define BLOCK_SIZE 32
#define RADIUS 5

__global__ void boxBlurFilter(unsigned char* input, unsigned char* output, int width, int height, int radius) {
    // Calcul des coordonn�es du thread en cours d'ex�cution
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // V�rification que les coordonn�es sont dans les limites de l'image
    if (index < width * height) {
        int x = index % width;
        int y = index / width;

        // Initialisation des variables de calcul du flou
        int sum = 0;
        int count = 0;

        // Parcours de la zone de flou
        for (int i = -radius; i <= radius; i++) {
            for (int j = -radius; j <= radius; j++) {
                // Calcul des coordonn�es du pixel � ajouter au calcul du flou
                int offsetX = x + i;
                int offsetY = y + j;

                // V�rification que les coordonn�es du pixel sont dans les limites de l'image
                if (offsetX >= 0 && offsetX < width && offsetY >= 0 && offsetY < height) {
                    // Ajout du pixel au calcul du flou
                    sum += input[offsetX + offsetY * width];
                    count++;
                }
            }
        }
        // Calcul de la valeur moyenne des pixels dans la zone de flou
        output[x + y * width] = sum / count;
    }
}

int main(int argc, char** argv) {
    // Verification de la pr�sence des deux arguments
    if (argc != 3) {
        printf("Usage: edge_detection <input_file> <output_file>\n");
        exit(-1);
    }

    // Charger l'image d'entr�e en niveaux de gris
    cv::Mat input = cv::imread(argv[1], cv::IMREAD_GRAYSCALE);
    if (input.empty()) {
        printf("Unable to load image '%s'\n", argv[1]);
        exit(-1);
    }

    // init cuda error
    hipError_t cudaStatus;
    hipError_t kernelStatus;

    // Allouer la m�moire pour les images d'entr�e et de sortie sur le GPU
    int width = input.cols;
    int height = input.rows;
    int radius = RADIUS;
    size_t inputSize = sizeof(unsigned char) * width * height;
    size_t outputSize = inputSize;
    unsigned char* d_input;
    unsigned char* d_output;
    cudaStatus = hipMalloc(&d_input, inputSize);
    if (cudaStatus != hipSuccess) {
        std::cout << "Error hipMalloc d_input: "  << std::endl;
        exit(-1);
    }
    cudaStatus = hipMalloc(&d_output, outputSize);
    if (cudaStatus != hipSuccess) {
        std::cout << "Error hipMalloc d_output: "  << std::endl;
        exit(-1);
    }

    // Creation des evenement pour le calcul du temps
    hipEvent_t start, stop;
    cudaStatus = hipEventCreate(&start);
    if (cudaStatus != hipSuccess) {
        std::cout << "Error hipEventCreate start: "  << std::endl;
        exit(-1);
    }
    cudaStatus = hipEventCreate(&stop);
    if (cudaStatus != hipSuccess) {
        std::cout << "Error hipEventCreate stop: " << std::endl;
        exit(-1);
    }

    // Mesure du temps de calcul du kernel uniquement
    hipEventRecord( start );

    // Copier l'image d'entr�e sur le GPU
    cudaStatus = hipMemcpy(d_input, input.data, inputSize, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        std::cout << "Error hipMemcpy - HostToDevice: "  << std::endl;
        exit(-1);
    }

    // D�finir la taille des blocs et des grilles
    dim3 blockSize(BLOCK_SIZE);
    dim3 gridSize((width*height + BLOCK_SIZE - 1) / BLOCK_SIZE);

    // Appliquer le filtre de flou sur l'image d'entr�e
    boxBlurFilter<<<gridSize, blockSize>>>(d_input, d_output, width, height, radius);
    kernelStatus = hipGetLastError();
    if (kernelStatus != hipSuccess) {
        std::cout << "CUDA Error: " << hipGetErrorString(kernelStatus) << std::endl;
        exit(-1);
    }

    // Copier l'image de sortie du GPU vers le CPU
    cv::Mat output(height, width, CV_8UC1);
    cudaStatus = hipMemcpy(output.data, d_output, outputSize, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        std::cout << "Error hipMemcpy - DeviceToHost: "  << std::endl;
        exit(-1);
    }

    // Fin du temps de calcul
    cudaStatus = hipEventRecord( stop );
    if (cudaStatus != hipSuccess) {
        std::cout << "Error hipEventRecord stop : "  << std::endl;
        exit(-1);
    }

    // Calcul du temps total
    cudaStatus = hipEventSynchronize(stop);
    if (cudaStatus != hipSuccess) {
        std::cout << "Error hipEventSynchronize stop : "  << std::endl;
        exit(-1);
    }
    float duration;
    hipEventElapsedTime( &duration, start, stop );
    std::cout << "time=" << duration << std::endl;

    hipEventDestroy(start);
    hipEventDestroy(stop);

    std::ofstream file("temps.txt", std::ios_base::app);
    file << "blur1D : " << duration << " ms\n" << std::endl;
    file.close();

    // Enregistrer l'image de sortie
    cv::imwrite(argv[2], output);

    // Lib�rer la m�moire allou�e sur le GPU
    hipFree(d_input);
    hipFree(d_output);
    
	return 0;
}