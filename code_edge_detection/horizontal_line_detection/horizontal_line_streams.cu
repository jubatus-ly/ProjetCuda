#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <vector>
#include <cmath>

#define BLOCK_SIZE 4
__global__ void horizontal_line_detection(unsigned char* input, unsigned char* output, int rows, int cols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row >= 1 && col >= 1 && row < rows - 1 && col < cols - 1) {
        int output_index = row * cols + col;

        int sum = 0;
        sum += -input[(row - 1) * cols + col - 1];
        sum += -input[(row - 1) * cols + col];
        sum += -input[(row - 1) * cols + col + 1];
        sum += 2 * input[row * cols + col - 1];
        sum += 2 * input[row * cols + col];
        sum += 2 * input[row * cols + col + 1];
        sum += -input[(row + 1) * cols + col - 1];
        sum += -input[(row + 1) * cols + col];
        sum += -input[(row + 1) * cols + col + 1];

        output[output_index] = (unsigned char) min(255,max(sum,0));
    }
}

int main(int argc, char** argv)
{
       
    if (argc != 3) {
        printf("Usage: edge_detection <input_file> <output_file>\n");
        exit(-1);
    }

    cv::Mat m_in = cv::imread(argv[1], cv::IMREAD_GRAYSCALE);

    if (m_in.empty()) {
        printf("Unable to load image '%s'\n", argv[1]);
        exit(-1);
    }
    auto gray = m_in.data;
    auto rows = m_in.rows;
    auto cols = m_in.cols;

    hipError_t cudaStatus;
    hipError_t kernelStatus;

    std::vector<unsigned char> conv(rows * cols);
    cv::Mat m_out(rows, cols, CV_8UC1, conv.data());

    unsigned char* gray_d;
    unsigned char* conv_d;

    cudaStatus = hipMalloc(&gray_d, rows * cols);
    if(cudaStatus != hipSuccess) {
        std::cout << "Error CudaMalloc gray_d" << std::endl;
        exit(-1);
    }

    cudaStatus = hipMalloc(&conv_d, rows * cols);
    if(cudaStatus != hipSuccess) {
        std::cout << "Error CudaMalloc conv_d: " <<std::endl;
        exit(-1);
    }


    dim3 t(BLOCK_SIZE, BLOCK_SIZE);
    dim3 b((cols - 1) / t.x + 1, (rows - 1) / t.y + 1);

    size_t inputSize = sizeof(unsigned char) * cols * rows;
    size_t outputSize = inputSize;
   
    hipEvent_t start, stop;
    cudaStatus = hipEventCreate(&start);
    if (cudaStatus != hipSuccess) {
        std::cout << "Error hipEventCreate start: "  << std::endl;
        exit(-1);
    }
    cudaStatus = hipEventCreate(&stop);
    if (cudaStatus != hipSuccess) {
        std::cout << "Error hipEventCreate stop: "  << std::endl;
        exit(-1);
    }

    
    cudaStatus = hipEventRecord(start);
    if (cudaStatus != hipSuccess) {
        std::cout << "Error hipEventRecord start: "  << std::endl;
        exit(-1);
    }

    
    hipStream_t stream1, stream2;
    cudaStatus = hipStreamCreate(&stream1);
    if (cudaStatus != hipSuccess) {
        std::cout << "Error hipStreamCreate stream1: "  << std::endl;
        exit(-1);
    }
    cudaStatus = hipStreamCreate(&stream2);
    if (cudaStatus != hipSuccess) {
        std::cout << "Error hipStreamCreate stream2: "  << std::endl;
        exit(-1);
    }

    cudaStatus = hipMemcpyAsync(gray_d, gray, inputSize / 2, hipMemcpyHostToDevice, stream1);
    if (cudaStatus != hipSuccess) {
        std::cout << "Error hipMemcpyAsync gray - HostToDevice: "  << std::endl;
        exit(-1);
    }
    
    cudaStatus = hipMemcpyAsync(gray_d + cols * (rows / 2), gray + cols * (rows / 2), inputSize / 2, hipMemcpyHostToDevice, stream2);
    if (cudaStatus != hipSuccess) {
        std::cout << "Error hipMemcpyAsync gray - HostToDevice: "  << std::endl;
        exit(-1);
    }
    
    cudaStatus = hipStreamSynchronize(stream1);
    if (cudaStatus != hipSuccess) {
        std::cout << "Error hipStreamSynchronize stream1: "  << std::endl;
        exit(-1);
    }    
    cudaStatus = hipStreamSynchronize(stream2);
    if (cudaStatus != hipSuccess) {
        std::cout << "Error hipStreamSynchronize stream2: "  << std::endl;
        exit(-1);
    }
    
    horizontal_line_detection<<< b, t, 0, stream1 >>>(gray_d, conv_d, rows/2, cols);
    kernelStatus = hipGetLastError();
    if (kernelStatus != hipSuccess) {
        std::cout << "CUDA Error: " << hipGetErrorString(kernelStatus) << std::endl;
        exit(-1);
    }
    
    horizontal_line_detection<<< b, t, 0, stream2 >>>(gray_d + cols * (rows / 2), conv_d + cols * (rows / 2), rows/2, cols);
    kernelStatus = hipGetLastError();
    if (kernelStatus != hipSuccess) {
        std::cout << "CUDA Error: " << hipGetErrorString(kernelStatus) << std::endl;
        exit(-1);
    }
    
    cudaStatus = hipStreamSynchronize(stream1);
    if (cudaStatus != hipSuccess) {
        std::cout << "Error hipStreamSynchronize stream1: "  << std::endl;
        exit(-1);
    }
    cudaStatus = hipStreamSynchronize(stream2);
    if (cudaStatus != hipSuccess) {
        std::cout << "Error hipStreamSynchronize stream2: "  << std::endl;
        exit(-1);
    }
    
    cv::Mat output(rows, cols, CV_8UC1);
    cudaStatus = hipMemcpyAsync(conv.data(), conv_d, outputSize, hipMemcpyDeviceToHost, stream1);
    if (cudaStatus != hipSuccess) {
        std::cout << "Error hipMemcpyAsync - DeviceToHost: "  << std::endl;
        exit(-1);
    }
    
    cudaStatus = hipEventRecord(stop);
    if (cudaStatus != hipSuccess) {
        std::cout << "Error hipEventRecord stop: " << std::endl;
        exit(-1);
    }
    
    
    cudaStatus = hipEventSynchronize(stop);
    if (cudaStatus != hipSuccess) {
        std::cout << "Error hipEventSynchronize: "  << std::endl;
        exit(-1);
    }
    float duration;
    cudaStatus = hipEventElapsedTime(&duration, start, stop);
    if (cudaStatus != hipSuccess) {
        std::cout << "Error hipEventElapsedTime: "  << std::endl;
        exit(-1);
    }
    std::cout << "time=" << duration << std::endl;

    cudaStatus=hipEventDestroy(start);
       if (cudaStatus != hipSuccess) 
  {
	std::cout << "Error CudaEvenDestroy start"  << " " ;
    exit(-1);

  }
    cudaStatus=hipEventDestroy(stop);
     if (cudaStatus != hipSuccess) 
  {
	std::cout << "Error CudaEvenDestroy stop"  << " " ;
    exit(-1);

  }

    std::ofstream file("temps.txt", std::ios_base::app);
    file << "blur : " << duration << " ms\n" << std::endl;
    file.close();

    cv::imwrite(argv[2], m_out);

    cudaStatus=hipFree(gray_d);
     if (cudaStatus != hipSuccess) 
  {
	std::cout << "Error Cudafree gray_d"  << " " ;
    exit(-1);
  }
    cudaStatus=hipFree(conv_d);
     if (cudaStatus != hipSuccess) 
  {
	std::cout << "Error Cudafree conv_d"  << " " ;
    exit(-1);
  }
    cudaStatus=hipStreamDestroy(stream1);
     if (cudaStatus != hipSuccess) 
  {
	std::cout << "Error CudaStreamDestroy stream1"  << " " ;
    exit(-1);

  }
    cudaStatus=hipStreamDestroy(stream2);
      if (cudaStatus != hipSuccess) 
  {
	std::cout << "Error CudaStreamDestroy stream1"  << " " ;
    exit(-1);

  }

    return 0;
}
