#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <opencv2/opencv.hpp>

#include <vector>
#include <chrono>
#include <fstream>

#define BLOCK_SIZE 32

__global__ void sobelFilter(unsigned char* input, unsigned char* output, int width, int height) {
    // Calcul des coordonn�es du thread en cours d'ex�cution
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // V�rification que les coordonn�es sont dans les limites de l'image
    if (x < width && y < height) {
        // Initialisation des variables de calcul du filtre
        int gx = 0, gy = 0;
        int i = x + y * width;

        // V�rification que les coordonn�es du pixel sont dans les limites de l'image
        if (x > 0 && y > 0 && x < width - 1 && y < height - 1) {
            // Calcule les valeurs des gradients gx et gy en utilisant l'op�rateur de Sobel
            gx = -1 * input[i - width - 1] + -2 * input[i - width] + -1 * input[i - width + 1] +
                 1 * input[i + width - 1] + 2 * input[i + width] + 1 * input[i + width + 1];
            gy = -1 * input[i - width - 1] + 1 * input[i - width + 1] +
                 -2 * input[i - 1] + 2 * input[i + 1] +
                 -1 * input[i + width - 1] + 1 * input[i + width + 1];
        }

        // Calcule la magnitude du gradient et stocke la valeur dans le tableau de sortie
        output[i] = (unsigned char) (__dsqrt_rn(gx * gx + gy * gy) / 4.0f);
    }
}

int main(int argc, char** argv) {
    // Verification de la pr�sence des deux arguments
    if (argc != 3) {
        printf("Usage: edge_detection <input_file> <output_file>\n");
        exit(-1);
    }

    // Charger l'image d'entr�e en niveaux de gris
    cv::Mat input = cv::imread(argv[1], cv::IMREAD_GRAYSCALE);
    if (input.empty()) {
        printf("Unable to load image '%s'\n", argv[1]);
        exit(-1);
    }

    // init cuda error
    hipError_t cudaStatus;
    hipError_t kernelStatus;

    // Allouer la m�moire pour les images d'entr�e et de sortie sur le GPU
    int width = input.cols;
    int height = input.rows;
    size_t inputSize = sizeof(unsigned char) * width * height;
    size_t outputSize = inputSize;
    unsigned char* d_input;
    unsigned char* d_output;
    cudaStatus = hipMalloc(&d_input, inputSize);
    if (cudaStatus != hipSuccess) {
        std::cout << "Error hipMalloc d_input: "  << std::endl;
        exit(-1);
    }
    cudaStatus = hipMalloc(&d_output, outputSize);
    if (cudaStatus != hipSuccess) {
        std::cout << "Error hipMalloc d_output: "  << std::endl;
        exit(-1);
    }

    // Creation des evenement pour le calcul du temps
    hipEvent_t start, stop;
    cudaStatus = hipEventCreate(&start);
    if (cudaStatus != hipSuccess) {
        std::cout << "Error hipEventCreate start: "  << std::endl;
        exit(-1);
    }
    cudaStatus = hipEventCreate(&stop);
    if (cudaStatus != hipSuccess) {
        std::cout << "Error hipEventCreate stop: " << std::endl;
        exit(-1);
    }

    // Mesure du temps de calcul du kernel uniquement
    hipEventRecord( start );

    // Copier l'image d'entr�e sur le GPU
    cudaStatus = hipMemcpy(d_input, input.data, inputSize, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        std::cout << "Error hipMemcpy - HostToDevice: "  << std::endl;
        exit(-1);
    }

    // D�finir la taille des blocs et des grilles
    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridSize((width + BLOCK_SIZE - 1) / BLOCK_SIZE, (height + BLOCK_SIZE - 1) / BLOCK_SIZE);

    // Appliquer le filtre de Sobel sur l'image d'entr�e pour d�tecter les bordures
    sobelFilter<<<gridSize, blockSize>>>(d_input, d_output, width, height);
    kernelStatus = hipGetLastError();
    if (kernelStatus != hipSuccess) {
        std::cout << "CUDA Error: " << hipGetErrorString(kernelStatus) << std::endl;
        exit(-1);
    }

    // Copier l'image de sortie du GPU vers le CPU
    cv::Mat output(height, width, CV_8UC1);
    cudaStatus = hipMemcpy(output.data, d_output, outputSize, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        std::cout << "Error hipMemcpy - DeviceToHost: "  << std::endl;
        exit(-1);
    }

    // Fin du temps de calcul
    cudaStatus = hipEventRecord( stop );
    if (cudaStatus != hipSuccess) {
        std::cout << "Error hipEventRecord stop : "  << std::endl;
        exit(-1);
    }

    // Calcul du temps total
    cudaStatus = hipEventSynchronize(stop);
    if (cudaStatus != hipSuccess) {
        std::cout << "Error hipEventSynchronize stop : "  << std::endl;
        exit(-1);
    }
    float duration;
    hipEventElapsedTime( &duration, start, stop );
    std::cout << "time=" << duration << std::endl;

    hipEventDestroy(start);
    hipEventDestroy(stop);

    std::ofstream file("temps.txt", std::ios_base::app);
    file << "sobel : " << duration << " ms\n" << std::endl;
    file.close();

    // Enregistrer l'image de sortie
    cv::imwrite(argv[2], output);

    // Lib�rer la m�moire allou�e sur le GPU
    hipFree(d_input);
    hipFree(d_output);
    
	return 0;
}