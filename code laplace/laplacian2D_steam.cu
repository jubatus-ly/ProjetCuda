#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <opencv2/opencv.hpp>

#include <vector>
#include <chrono>
#include <fstream>


__global__ void laplacianOperator(unsigned char* input, unsigned char* output, int rows, int cols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row > 0 && col > 0 && row < rows - 1 && col < cols - 1) {
        int output_index = row * cols + col;

        int sum = 0;
        sum += -input[(row - 1) * cols + col - 1];
        sum += -input[(row - 1) * cols + col];
        sum += -input[(row - 1) * cols + col + 1];
        sum += -input[row * cols + col - 1];
        sum += 8 * input[row * cols + col];
        sum += -input[row * cols + col + 1];
        sum += -input[(row + 1) * cols + col - 1];
        sum += -input[(row + 1) * cols + col];
        sum += -input[(row + 1) * cols + col + 1];

        output[output_index] = (unsigned char) min(255,max(sum,0));
    }
}


int main(int argc, char** argv)
{
    if (argc != 3) {
        printf("Usage: ./programme <input_file> <output_file>\n");
        exit(-1);
    }

    cv::Mat m_in = cv::imread(argv[1], cv::IMREAD_GRAYSCALE);
    if (m_in.empty()) {
        printf("Unable to load image '%s'\n", argv[1]);
        exit(-1);
    }

    auto rows = m_in.rows;
    auto cols = m_in.cols;

    hipError_t cudaStatus;
    hipError_t kernelStatus;

    std::vector<unsigned char> lap(rows * cols);
    cv::Mat m_out(rows, cols, CV_8UC1, lap.data());

    unsigned char* gray_d;
    unsigned char* lap_d;

    cudaStatus = hipMalloc(&gray_d, rows * cols);
    if (cudaStatus != hipSuccess) {
        std::cout << "Error CudaMalloc gray_d: "  << std::endl;
        exit(-1);
    }

    cudaStatus = hipMalloc(&lap_d, rows * cols);
    if (cudaStatus != hipSuccess) {
        std::cout << "Error CudaMalloc lap_d: "  << std::endl;
        exit(-1);
    }

    dim3 t(32, 32);
    dim3 b((cols - 1) / t.x + 1, (rows - 1) / t.y + 1);

    hipStream_t stream1, stream2;
    cudaStatus = hipStreamCreate(&stream1);
    if (cudaStatus != hipSuccess) {
        std::cout << "Error hipStreamCreate stream1: "  << std::endl;
        exit(-1);
    }

    cudaStatus = hipStreamCreate(&stream2);
    if (cudaStatus != hipSuccess) {
        std::cout << "Error hipStreamCreate stream2: "  << std::endl;
        exit(-1);
    }

    hipEvent_t start, stop;
    cudaStatus = hipEventCreate(&start);
    if (cudaStatus != hipSuccess) {
        std::cout << "Error hipEventCreate start: "  << std::endl;
        exit(-1);
    }

    cudaStatus = hipEventCreate(&stop);
    if (cudaStatus != hipSuccess) {
        std::cout << "Error hipEventCreate stop: "  << std::endl;
        exit(-1);
    }

    const int size = rows * cols;
    const int half_size = size / 2;

    cudaStatus = hipMemcpyAsync(gray_d, m_in.data, half_size, hipMemcpyHostToDevice, stream1);
    if (cudaStatus != hipSuccess) {
        std::cout << "Error hipMemcpyAsync gray - HostToDevice: "  << std::endl;
        exit(-1);
    }

    cudaStatus = hipMemcpyAsync(gray_d + half_size, m_in.data + half_size, size - half_size, hipMemcpyHostToDevice, stream2);
    if (cudaStatus != hipSuccess) {
        std::cout << "Error hipMemcpyAsync gray - HostToDevice: "  << std::endl;
        exit(-1);
    }

    cudaStatus = hipEventRecord(start);
    if (cudaStatus != hipSuccess) {
        std::cout << "Error hipEventRecord start: "  << std::endl;
        exit(-1);
    }

    laplacianOperator<<< b, t, 0, stream1 >>>(gray_d, lap_d, rows / 2 + 1, cols);
    kernelStatus = hipGetLastError();
    if (kernelStatus != hipSuccess) {
        std::cout << "CUDA Error: " << hipGetErrorString(kernelStatus) << std::endl;
        exit(-1);
    }

    laplacianOperator<<< b, t, 0, stream2 >>>(gray_d + (rows / 2 - 1) * cols, lap_d + (rows / 2 - 1) * cols, rows - rows / 2 + 1, cols);
    kernelStatus = hipGetLastError();
    if (kernelStatus != hipSuccess) {
        std::cout << "CUDA Error: " << hipGetErrorString(kernelStatus) << std::endl;
        exit(-1);
    }

    cudaStatus = hipEventRecord(stop);
    if (cudaStatus != hipSuccess) {
        std::cout << "Error hipEventRecord stop: " << std::endl;
        exit(-1);
    }

    cudaStatus = hipMemcpyAsync(lap.data(), lap_d, half_size, hipMemcpyDeviceToHost, stream1);
    if (cudaStatus != hipSuccess) {
        std::cout << "Error hipMemcpyAsync lap - DeviceToHost: "  << std::endl;
        exit(-1);
    }

    cudaStatus = hipMemcpyAsync(lap.data() + half_size, lap_d + half_size, size - half_size, hipMemcpyDeviceToHost, stream2);
    if (cudaStatus != hipSuccess) {
        std::cout << "Error hipMemcpyAsync lap - DeviceToHost: "  << std::endl;
        exit(-1);
    }

    cudaStatus = hipStreamSynchronize(stream1);
    if (cudaStatus != hipSuccess) {
        std::cout << "Error hipStreamSynchronize stream1: "  << std::endl;
        exit(-1);
    }

    cudaStatus = hipStreamSynchronize(stream2);
    if (cudaStatus != hipSuccess) {
        std::cout << "Error hipStreamSynchronize stream2: "  << std::endl;
        exit(-1);
    }

    cudaStatus = hipEventSynchronize(stop);
    if (cudaStatus != hipSuccess) {
        std::cout << "Error hipEventSynchronize: "  << std::endl;
        exit(-1);
    }

    float duration;
    cudaStatus = hipEventElapsedTime(&duration, start, stop);
    if (cudaStatus != hipSuccess) {
        std::cout << "Error hipEventElapsedTime: "  << std::endl;
        exit(-1);
    }

    std::cout << "time=" << duration << std::endl;

    cv::imwrite(argv[2], m_out);

      cudaStatus=hipStreamDestroy(stream1);
     if (cudaStatus != hipSuccess) 
  {
	std::cout << "Error CudaStreamDestroy stream1"  << " " ;
    exit(-1);

  }

    cudaStatus=hipStreamDestroy(stream2);
      if (cudaStatus != hipSuccess) 
  {
	std::cout << "Error CudaStreamDestroy stream1"  << " " ;
    exit(-1);

  }

    cudaStatus=hipEventDestroy(start);
       if (cudaStatus != hipSuccess) 
  {
	std::cout << "Error CudaEvenDestroy start"  << " " ;
    exit(-1);

  }
   
    cudaStatus=hipEventDestroy(stop);
     if (cudaStatus != hipSuccess) 
  {
	std::cout << "Error CudaEvenDestroy stop"  << " " ;
    exit(-1);

  }

    cv::imwrite(argv[2], m_out);

    cudaStatus=hipFree(gray_d);
     if (cudaStatus != hipSuccess) 
  {
	std::cout << "Error Cudafree gray_d"  << " " ;
    exit(-1);
  }
    cudaStatus=hipFree(lap_d);
    if (cudaStatus != hipSuccess) 
  {
	std::cout << "Error Cudafree lap_d"  << " " ;
    exit(-1);
  }


    return 0;
}
