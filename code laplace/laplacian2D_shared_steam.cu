#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <opencv2/opencv.hpp>

#include <vector>
#include <chrono>
#include <fstream>

__global__ void laplacianOperatorShared(unsigned char* input, unsigned char* output, int rows, int cols) {
    int row = blockIdx.y * (blockDim.y - 2) + threadIdx.y - 1;
    int col = blockIdx.x * (blockDim.x - 2) + threadIdx.x - 1;

    extern __shared__ unsigned char sh[];

    int sh_row = threadIdx.y;
    int sh_col = threadIdx.x;

    int sh_index = sh_row * (blockDim.x + 2) + sh_col;

    if (row >= 0 && col >= 0 && row < rows && col < cols) {
        sh[sh_index] = input[row * cols + col];
    } else {
        sh[sh_index] = 0;
    }

    __syncthreads();

    if (threadIdx.y > 0 && threadIdx.x > 0 && threadIdx.y < blockDim.y - 1 && threadIdx.x < blockDim.x - 1 && row >= 1 && col >= 1 && row < rows - 1 && col < cols - 1) {
        int output_index = row * cols + col;

        int sum = 0;
        sum += -sh[(sh_row - 1) * (blockDim.x + 2) + sh_col - 1];
        sum += -sh[(sh_row - 1) * (blockDim.x + 2) + sh_col];
        sum += -sh[(sh_row - 1) * (blockDim.x + 2) + sh_col + 1];
        sum += -sh[sh_row * (blockDim.x + 2) + sh_col - 1];
        sum += 8 * sh[sh_row * (blockDim.x + 2) + sh_col];
        sum += -sh[sh_row * (blockDim.x + 2) + sh_col + 1];
        sum += -sh[(sh_row + 1) * (blockDim.x + 2) + sh_col - 1];
        sum += -sh[(sh_row + 1) * (blockDim.x + 2) + sh_col];
        sum += -sh[(sh_row + 1) * (blockDim.x + 2) + sh_col + 1];

        output[output_index] = (unsigned char)min(255, max(sum, 0));
    }
}

int main(int argc, char** argv)
{
    if (argc != 3) {
        printf("Usage: ./programme <input_file> <output_file>\n");
        exit(-1);
    }

    cv::Mat m_in = cv::imread(argv[1], cv::IMREAD_GRAYSCALE);
    if (m_in.empty()) {
        printf("Unable to load image '%s'\n", argv[1]);
        exit(-1);
    }

    auto rows = m_in.rows;
    auto cols = m_in.cols;

    hipError_t cudaStatus;
    hipError_t kernelStatus;

    std::vector<unsigned char> lap(rows * cols);
    cv::Mat m_out(rows, cols, CV_8UC1, lap.data());

    unsigned char* gray_d;
    unsigned char* lap_d;

    cudaStatus = hipMalloc(&gray_d, rows * cols);
    if (cudaStatus != hipSuccess) {
        std::cout << "Error CudaMalloc gray_d: " << std::endl;
        exit(-1);
    }

    cudaStatus = hipMalloc(&lap_d, rows * cols);
    if (cudaStatus != hipSuccess) {
        std::cout << "Error CudaMalloc lap_d: " <<  std::endl;
        exit(-1);
    }

    dim3 block(32, 32);
    dim3 grid((cols - 1) / (block.x - 2) + 1, (rows - 1) / (block.y - 2) + 1);

    hipStream_t stream1, stream2;
    cudaStatus = hipStreamCreate(&stream1);
    if (cudaStatus != hipSuccess) {
        std::cout << "Error hipStreamCreate stream1: "  << std::endl;
        exit(-1);
    }

    cudaStatus = hipStreamCreate(&stream2);
    if (cudaStatus != hipSuccess) {
        std::cout << "Error hipStreamCreate stream2: "  << std::endl;
        exit(-1);
    }

    hipEvent_t start, stop;

    cudaStatus = hipEventCreate(&start);
    if (cudaStatus != hipSuccess) {
        std::cout << "Error hipEventCreate start: "  << std::endl;
        exit(-1);
    }

    cudaStatus = hipEventCreate(&stop);
    if (cudaStatus != hipSuccess) {
        std::cout << "Error hipEventCreate stop: " << std::endl;
        exit(-1);
    }

    const int size = rows * cols;
    const int half_size = size / 2;

    cudaStatus = hipMemcpyAsync(gray_d, m_in.data, half_size, hipMemcpyHostToDevice, stream1);
    if (cudaStatus != hipSuccess) {
        std::cout << "Error hipMemcpyAsync gray - HostToDevice: "  << std::endl;
        exit(-1);
    }

    cudaStatus = hipMemcpyAsync(gray_d + half_size, m_in.data + half_size, size - half_size, hipMemcpyHostToDevice, stream2);
    if (cudaStatus != hipSuccess) {
        std::cout << "Error hipMemcpyAsync gray - HostToDevice: "  << std::endl;
        exit(-1);
    }

    cudaStatus = hipEventRecord(start);
    if (cudaStatus != hipSuccess) {
        std::cout << "Error hipEventRecord start: "  << std::endl;
        exit(-1);
    }

    laplacianOperatorShared<<<grid, block, (block.x + 2) * (block.y + 2) * sizeof(unsigned char), stream1>>>(gray_d, lap_d, rows / 2 + 1, cols);
    kernelStatus = hipGetLastError();
    if (kernelStatus != hipSuccess) {
        std::cout << "CUDA Error: " << hipGetErrorString(kernelStatus) << std::endl;
        exit(-1);
    }

    laplacianOperatorShared<<<grid, block, (block.x + 2) * (block.y + 2) * sizeof(unsigned char), stream2>>>(gray_d + (rows / 2 - 1) * cols, lap_d + (rows / 2 - 1) * cols, rows - rows / 2 + 1, cols);
    kernelStatus = hipGetLastError();
    if (kernelStatus != hipSuccess) {
        std::cout << "CUDA Error: " << hipGetErrorString(kernelStatus) << std::endl;
        exit(-1);
    }

    cudaStatus = hipEventRecord(stop);
    if (cudaStatus != hipSuccess) {
        std::cout << "Error hipEventRecord stop: "  << std::endl;
        exit(-1);
    }

    cudaStatus = hipMemcpyAsync(lap.data(), lap_d, half_size, hipMemcpyDeviceToHost, stream1);
    if (cudaStatus != hipSuccess) {
        std::cout << "Error hipMemcpyAsync lap - DeviceToHost: "  << std::endl;
        exit(-1);
    }

    cudaStatus = hipMemcpyAsync(lap.data() + half_size, lap_d + half_size, size - half_size, hipMemcpyDeviceToHost, stream2);
    if (cudaStatus != hipSuccess) {
        std::cout << "Error hipMemcpyAsync lap - DeviceToHost: "  << std::endl;
        exit(-1);
    }

    cudaStatus = hipStreamSynchronize(stream1);
    if (cudaStatus != hipSuccess) {
        std::cout << "Error hipStreamSynchronize stream1: " << std::endl;
        exit(-1);
    }

    cudaStatus = hipStreamSynchronize(stream2);
    if (cudaStatus != hipSuccess) {
        std::cout << "Error hipStreamSynchronize stream2: "  << std::endl;
        exit(-1);
    }

    cudaStatus = hipEventSynchronize(stop);
    if (cudaStatus != hipSuccess) {
        std::cout << "Error hipEventSynchronize stop : "  << std::endl;
        exit(-1);
    }

    float duration;
    cudaStatus = hipEventElapsedTime(&duration, start, stop);
    if (cudaStatus != hipSuccess) {
        std::cout << "Error hipEventElapsedTime: "<< std::endl;
        exit(-1);
    }

    std::cout << "time=" << duration << std::endl;

    cv::imwrite(argv[2], m_out);

    

    cudaStatus=hipStreamDestroy(stream1);
     if (cudaStatus != hipSuccess) 
  {
	std::cout << "Error CudaStreamDestroy stream1"  << " " ;
    exit(-1);

  }

    cudaStatus=hipStreamDestroy(stream2);
      if (cudaStatus != hipSuccess) 
  {
	std::cout << "Error CudaStreamDestroy stream1"  << " " ;
    exit(-1);

  }

    cudaStatus=hipEventDestroy(start);
       if (cudaStatus != hipSuccess) 
  {
	std::cout << "Error CudaEvenDestroy start"  << " " ;
    exit(-1);

  }
   
    cudaStatus=hipEventDestroy(stop);
     if (cudaStatus != hipSuccess) 
  {
	std::cout << "Error CudaEvenDestroy stop"  << " " ;
    exit(-1);

  }

    cv::imwrite(argv[2], m_out);

    cudaStatus=hipFree(gray_d);
     if (cudaStatus != hipSuccess) 
  {
	std::cout << "Error Cudafree gray_d"  << " " ;
    exit(-1);
  }
    cudaStatus=hipFree(lap_d);
    if (cudaStatus != hipSuccess) 
  {
	std::cout << "Error Cudafree lap_d"  << " " ;
    exit(-1);

  }

    return 0;
}
