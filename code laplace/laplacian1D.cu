#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <opencv2/opencv.hpp>

#include <vector>
#include <chrono>
#include <fstream>


__global__ void laplacianOperator(unsigned char* input, unsigned char* output, int rows, int cols) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    int row = idx / cols;
    int col = idx % cols;

    if (row > 0 && col > 0 && row < rows - 1 && col < cols - 1) {
        int output_index = row * cols + col;

        int sum = 0;
        sum += -input[(row - 1) * cols + col - 1];
        sum += -input[(row - 1) * cols + col];
        sum += -input[(row - 1) * cols + col + 1];
        sum += -input[row * cols + col - 1];
        sum += 8 * input[row * cols + col];
        sum += -input[row * cols + col + 1];
        sum += -input[(row + 1) * cols + col - 1];
        sum += -input[(row + 1) * cols + col];
        sum += -input[(row + 1) * cols + col + 1];

        output[output_index] = (unsigned char) min(255,max(sum,0));
    }
}

int main(int argc, char** argv)
{   
     if (argc != 3) {
        printf("Usage: ./programme <input_file> <output_file>\n");
        exit(-1);
    }


    cv::Mat m_in = cv::imread(argv[1], cv::IMREAD_GRAYSCALE);
      if (m_in.empty()) {
        printf("Unable to load image '%s'\n", argv[1]);
        exit(-1);
    }


    auto gray = m_in.data;
    auto rows = m_in.rows;
    auto cols = m_in.cols;
    hipError_t cudaStatus;
    hipError_t kernelStatus;

    std::vector<unsigned char> lap(rows * cols);
    cv::Mat m_out(rows, cols, CV_8UC1, lap.data());

    unsigned char* gray_d;
    unsigned char* lap_d;

    cudaStatus=hipMalloc(&gray_d, rows * cols);
    if (cudaStatus != hipSuccess)
    {
        std::cout << "Error CudaMalloc gray_d"  << " ";
        exit(-1);

    }


    cudaStatus=hipMalloc(&lap_d, rows * cols);
    if (cudaStatus != hipSuccess)
  {
	std::cout << "Error CudaMalloc lap_d"  << " ";
    exit(-1);

  }



    int blockSize = 4;
    int gridSize = (rows * cols + blockSize - 1) / blockSize;

    // Creation des evenement pour le calcul du temps
    hipEvent_t start, stop;

    cudaStatus=hipEventCreate( &start );
     if (cudaStatus  != hipSuccess)
  {
	  std::cout << "Error Eventcreate start" << " ";
      exit(-1);

  }
    cudaStatus=hipEventCreate( &stop );
     if (cudaStatus  != hipSuccess)
  {
	  std::cout << "Error Eventcreate stop" << " ";
      exit(-1);

    
  }

    cudaStatus=hipMemcpy(gray_d, gray, rows * cols, hipMemcpyHostToDevice);
    if (cudaStatus  != hipSuccess)
  {
	  std::cout << "Error hipMemcpy gray - HostToDevice" << " ";
      exit(-1);

  }


    cudaStatus=hipEventRecord(start);
    if (cudaStatus  != hipSuccess)
  {
	  std::cout << "Error cudaStart" << " ";
      exit(-1);

  }



    laplacianOperator<<<gridSize, blockSize>>>(gray_d, lap_d, rows, cols);
    kernelStatus = hipGetLastError();
   if ( kernelStatus != hipSuccess )
   {
	   std::cout << "CUDA Error"<< hipGetErrorString(kernelStatus) << " ";
       exit(-1);

  }

    // End of computation time
   cudaStatus= hipEventRecord(stop);
   if (cudaStatus  != hipSuccess)
  {
	  std::cout << "Error cudaStop" << " ";
      exit(-1);

  }


   cudaStatus= hipMemcpy(lap.data(), lap_d, rows * cols, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) 
  {
	std::cout << "Error Cuda Memcpy lap DeviceToHost"  << " " ;
     exit(-1);

  }

    // Calcul du temps total
   cudaStatus= hipEventSynchronize( stop );
      if (cudaStatus != hipSuccess) 
  {
	std::cout << "Error CudaEvenSychronize"  << " " ;
    exit(-1);

  }


    float duration;
    hipEventElapsedTime( &duration, start, stop );
    std::cout << "time=" << duration << std::endl;

   cudaStatus=hipEventDestroy(start);
       if (cudaStatus != hipSuccess) 
  {
	std::cout << "Error CudaEvenDestroy start"  << " " ;
    exit(-1);

  }
   
    cudaStatus=hipEventDestroy(stop);
     if (cudaStatus != hipSuccess) 
  {
	std::cout << "Error CudaEvenDestroy stop"  << " " ;
    exit(-1);

  }

    cv::imwrite(argv[2], m_out);

    cudaStatus=hipFree(gray_d);
     if (cudaStatus != hipSuccess) 
  {
	std::cout << "Error Cudafree gray_d"  << " " ;
    exit(-1);
  }
    cudaStatus=hipFree(lap_d);
    if (cudaStatus != hipSuccess) 
  {
	std::cout << "Error Cudafree lap_d"  << " " ;
    exit(-1);

  }

    return 0;
}
